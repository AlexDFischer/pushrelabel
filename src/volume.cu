#include "hip/hip_runtime.h"
#include "project.h"
#include "volume.h"

__host__ __device__
unsigned long getIntensity(Volume *volume, int x, int y, int z)
{
  size_t index = z * volume->width * volume->height + y * volume->width + x;
  switch (volume->bytesPerPixel)
  {
    case 1:
      return ((uint8_t *) volume->data)[index];
    case 2:
      return ((uint16_t *) volume->data)[index];
    case 4:
      return ((uint32_t *) volume->data)[index];
    case 8:
      return ((uint64_t *) volume->data)[index];
  }
  // should never get to here
  return 0;
}

__host__ __device__
void setIntensity(Volume *volume, int x, int y, int z, unsigned long intensity)
{
  size_t index = z * volume->width * volume->height + y * volume->width + x;
  switch (volume->bytesPerPixel)
  {
    case 1:
      ((uint8_t *) (volume->data))[index] = intensity;
      break;
    case 2:
      ((uint16_t *) (volume->data))[index] = intensity;
      break;
    case 4:
      ((uint32_t *) (volume->data))[index] = intensity;
      break;
    case 8:
      ((uint64_t *) (volume->data))[index] = intensity;
      break;
  }
}

/**
 * Calls hipMallocManaged on the data of the given volume, assuming the width,
 * height, and depth of the volume have been set already.
 */
void cudaMallocManagedVolume(Volume *volume)
{
  hipMallocManaged(&(volume->data), volume->width * volume->height * volume->depth * volume->bytesPerPixel);
}

void mallocVolume(Volume *volume)
{
  volume->data = (char *) malloc(volume->width * volume->height * volume->depth * volume->bytesPerPixel);
}

unsigned long maxIntensity(Volume *volume)
{
  int x, y, z;
  unsigned long max = getIntensity(volume, 0, 0, 0);
  for (z = 0; z < volume->depth; z++)
  {
    for (y = 0; y < volume->height; y++)
    {
      for (x = 0; x < volume->width; x++)
      {
        unsigned long val = getIntensity(volume, x, y, z);
        if (val > max)
        {
          max = val;
        }
      }
    }
  }
  return max;
}

unsigned long minIntensity(Volume *volume)
{
  int x, y, z;
  unsigned long min = getIntensity(volume, 0, 0, 0);
  for (z = 0; z < volume->depth; z++)
  {
    for (y = 0; y < volume->height; y++)
    {
      for (x = 0; x < volume->width; x++)
      {
        unsigned long val = getIntensity(volume, x, y, z);
        if (val < min)
        {
          min = val;
        }
      }
    }
  }
  return min;
}

void printVolume(Volume *volume)
{
  int x, y, z;
  for (z = 0; z < volume->depth; z++)
  {
    printf("SLICE %d\n", z);
    for (y = 0; y < volume->height; y++)
    {
      for (x = 0; x < volume->width; x++)
      {
        printf("%02lx ", getIntensity(volume, x, y, z));
      }
      printf("\n");
    }
  }
}

/**
 * Reads the given RAW file into the given volume. volume->data is
 * hipMallocManaged and should be cudaFreed when done. fileName should not have
 * an extension, as the RAW and TXT extensions will be added onto it.
 */
int readRaw(Volume **volume, char *fileName)
{
  hipError_t r = hipMallocManaged(volume, sizeof(Volume));
  if (r != hipSuccess)
  {
    fprintf(stderr, "%s: unable to hipMallocManaged: %d: %s\n", programName, (int) r, hipGetErrorString(r));
    exit(-1);
  }
  int len = strlen(fileName);
  char *fileNameExt = (char *) malloc(len + 5);
  strcpy(fileNameExt, fileName);
  strcpy(fileNameExt + len, ".txt");
  FILE *f = fopen(fileNameExt, "r");
  if (f == NULL)
  {
    fprintf(stderr, "%s: unable to open file %s: %s\n", programName, fileNameExt, strerror(errno));
    return -1;
  }
  if (fscanf(f, "%dx%dx%d\n", &((**volume).width), &((**volume).height), &((**volume).depth)) != 3)
  {
    fprintf(stderr, "%s: invalid first line of %s\n", programName, fileNameExt);
    return -1;
  }
  if (fscanf(f, "%d\n", &((**volume).bytesPerPixel)) != 1)
  {
    fprintf(stderr, "%s: invalid second line of %s\n", programName, fileNameExt);
    return -1;
  }
  int scaleX, scaleY, scaleZ;
  if (fscanf(f, "scale: %d:%d:%d", &scaleX, &scaleY, &scaleZ) != 3)
  {
    fprintf(stderr, "%s: invalid third line of %s\n", programName, fileNameExt);
    return -1;
  }
  if (fclose(f))
  {
    fprintf(stderr, "%s: unable to close file %s: %s\n", programName, fileNameExt, strerror(errno));
    return -1;
  }
  printf("dealing with a %dx%dx%d volume with %d bytes per pixel\n", (**volume).width, (**volume).height, (**volume).depth, (**volume).bytesPerPixel);
  cudaMallocManagedVolume(*volume);
  strcpy(fileNameExt + len, ".raw");
  f = fopen(fileNameExt, "r");
  fread((**volume).data, (**volume).bytesPerPixel, (**volume).width * (**volume).height * (**volume).depth, f);
  if (ferror(f))
  {
    fprintf(stderr, "%s: error reading from file %s: %s\n", programName, fileNameExt, strerror(ferror(f)));
    return -1;
  }
  free(fileNameExt);
  return 0;
}

/**
 * Writes the given volume to the given RAW file. fileName should not have an
 * extension, as the RAW and TXT extensions will be added onto it.
 */
int writeRaw(Volume *volume, char *fileName)
{
  int len = strlen(fileName);
  char *fileNameExt = (char *) malloc(len + 5);
  strcpy(fileNameExt, fileName);
  strcpy(fileNameExt + len, ".txt");
  FILE *f = fopen(fileNameExt, "w");
  if (f == NULL)
  {
    fprintf(stderr, "%s: unable to open file %s: %s\n", programName, fileNameExt, strerror(errno));
    return -1;
  }
  fprintf(f, "%dx%dx%d\n", volume->width, volume->height, volume->depth);
  fprintf(f, "%d\n", volume->bytesPerPixel);
  fprintf(f, "scale: 1:1:1\n");
  if (ferror(f))
  {
    fprintf(stderr, "%s: error writing to file %s: %s\n", programName, fileNameExt, strerror(ferror(f)));
    return -1;
  }
  fclose(f);
  strcpy(fileNameExt + len, ".raw");
  f = fopen(fileNameExt, "w");
  if (f == NULL)
  {
    fprintf(stderr, "%s: unable to open file %s: %s\n", programName, fileNameExt, strerror(errno));
    return -1;
  }
  fwrite(volume->data, volume->bytesPerPixel, volume->width * volume->height * volume->depth, f);
  if (ferror(f))
  {
    fprintf(stderr, "%s: error writing to file %s: %s\n", programName, fileNameExt, strerror(ferror(f)));
    return -1;
  }
  fclose(f);
  free(fileNameExt);
  return 0;
}

/**
 * Writes the given volume to the given tiff directory, creating it if the
 * durectory doesn't already exist. Returns 0 on success, -1 on error.
 * TODO this method doesn't work for some reason
 */
int writeTiff(Volume *volume, char *dirName)
{
  // if the directory doesn't exist, make it
  struct stat st = {0};
  if (stat(dirName, &st) == -1)
  {
      if (mkdir(dirName, 0700) != 0)
      {
        fprintf(stderr, "%s: unable to create directory %s: %s\n", programName, dirName, strerror(errno));
        return -1;
      }
  }
  int numberLength = (int) ceil(log(volume->depth) / log(10));
  char fileName[strlen(dirName) + 1 + numberLength + 5];
  TIFF *tif;
  int z, y;
  unsigned char *data = (unsigned char *) volume->data;
  unsigned char *buf = (unsigned char *) _TIFFmalloc(volume->width);
  for (z = 0; z < volume->depth;z++)
  {
    sprintf(fileName, "%s/%d.tif", dirName, z);
    if ((tif = TIFFOpen(fileName, "w")) == NULL)
    {
      fprintf(stderr, "%s: unable to open file %s\n", programName, fileName);
      return -1;
    }
    TIFFSetField(tif, TIFFTAG_IMAGEWIDTH, volume->width);
    TIFFSetField(tif, TIFFTAG_IMAGELENGTH, volume->height);
    TIFFSetField(tif, TIFFTAG_SAMPLESPERPIXEL, 1);
    TIFFSetField(tif, TIFFTAG_BITSPERSAMPLE, 8);
    TIFFSetField(tif, TIFFTAG_ROWSPERSTRIP, 1);
    for (y = 0; y < volume->height; y++)
    {
      memcpy(buf, volume->data, volume->width);
      data += volume->width;
      if (TIFFWriteScanline(tif, buf, y) != 1)
      {
        fprintf(stderr, "%s: error writing to %s\n", programName, fileName);
        _TIFFfree(buf);
        return -1;
      }
    }
    TIFFClose(tif);
  }
  _TIFFfree(buf);
  return 0;
}
